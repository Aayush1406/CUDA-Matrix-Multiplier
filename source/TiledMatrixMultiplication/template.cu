#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <wb.h>

#define TILE_WIDTH 16 	//do not change this value

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
// Compute C = A * B
// Compute C = A * B
__global__ void matrixMultiplyShared(float* A, float* B, float* C, int numARows, int numAColumns, int numBColumns) {
    //@@ Insert code to implement tiled matrix multiplication here
    //@@ You have to use shared memory to write this kernel

    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;

    int row = blockDim.y * by + ty;
    int col = blockDim.x * bx + tx;

    float pvalue = 0;
    
    for (int p = 0; p < numAColumns / TILE_WIDTH; p++) { // Will run 128 times

        ds_A[ty][tx] =  A[row * numAColumns + p*TILE_WIDTH+tx];
        ds_B[ty][tx] =  B[(p*TILE_WIDTH+ty) * numBColumns + col];
        
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++) {

        pvalue = pvalue + ds_A[ty][i] * ds_B[i][tx];

        __syncthreads();
        }

    }

    C[row * numARows + col] = pvalue;
}

//    float Pvalue = 0;
//    // Loop over the M and N tiles required to compute the P element
//    for (int p = 0; p < Width / TILE_WIDTH; ++p) {
//        // Collaborative loading of M and N tiles into shared memory
//        ds_M[ty][tx] = M[Row * Width + p * TILE_WIDTH + tx];
//        ds_N[ty][tx] = N[(p * TILE_WIDTH + ty) * Width + Col];
//        __syncthreads();
//        for (int i = 0; i < TILE_WIDTH; ++i)Pvalue += ds_M[ty][i] * ds_N[i][tx];
//        __synchthreads();
//    }
//    P[Row * Width + Col] = Pvalue;
//}


int main(int argc, char** argv) {
    wbArg_t args;
    float* hostA; // The A matrix
    float* hostB; // The B matrix
    float* hostC; // The output C matrix
    float* deviceA;
    float* deviceB;
    float* deviceC;
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows;    // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set
                     // this)

    hostC = NULL;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float*)wbImport(wbArg_getInputFile(args, 0), &numARows,
        &numAColumns);
    hostB = (float*)wbImport(wbArg_getInputFile(args, 1), &numBRows,
        &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;      // Rows of C = Rows of A
    numCColumns = numBColumns; // Columns of C = Columns of B

    //@@ Allocate the hostC matrix
    hostC = (float*)malloc(numCRows * numCColumns * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc((void**)&deviceA, numARows * numAColumns * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceB, numBRows * numBColumns * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceC, numCRows * numCColumns * sizeof(float)));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((numCColumns + TILE_WIDTH - 1) / TILE_WIDTH, (numCRows + TILE_WIDTH - 1) / TILE_WIDTH);


    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiplyShared << <dimGrid, dimBlock >> > (deviceA, deviceB, deviceC, numARows, numAColumns, numBColumns);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree(deviceA));
    wbCheck(hipFree(deviceB));
    wbCheck(hipFree(deviceC));

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
