#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <wb.h>

#define TILE_WIDTH 16 	//do not change this value

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
// Compute C = A * B
// Compute C = A * B
__global__ void matrixMultiplyShared(float* A, float* B, float* C, int numARows, int numAColumns, int numBColumns) {
    //@@ Insert code to implement tiled matrix multiplication here
    //@@ You have to use shared memory to write this kernel

    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;

    int row = blockDim.y * by + ty;
    int col = blockDim.x * bx + tx;

    float pvalue = 0;
    
    for (int p = 0; p < (numAColumns -1 )/ TILE_WIDTH + 1; p++) { // Will run 128 times

        if (row < numARows && p * TILE_WIDTH + tx < numAColumns) {

            ds_A[ty][tx] = A[row * numAColumns + p * TILE_WIDTH + tx];
        }
        else {
            ds_A[ty][tx] = 0.0;

        }


        if (col < numBColumns && p * TILE_WIDTH + ty < numAColumns) {

           ds_B[ty][tx] = B[(p * TILE_WIDTH + ty) * numBColumns + col];

        }
        else {

            ds_B[ty][tx] = 0.0;

        }

        __syncthreads();
        for (int i = 0; i < TILE_WIDTH; i++) {

        pvalue = pvalue + ds_A[ty][i] * ds_B[i][tx];

        __syncthreads();
        }

    }

    if (row < numARows && col < numBColumns) {

        C[row * numBColumns + col] = pvalue;

    }
}

//__global__ void matrixMultiplyShared(float* A, float* B, float* C,
//    int numARows, int numAColumns,
//    int numBColumns) {
//    //@@ Insert code to implement tiled matrix multiplication here
//    //@@ You have to use shared memory to write this kernel
//    __shared__ float Ashared[TILE_WIDTH][TILE_WIDTH];
//    __shared__ float Bshared[TILE_WIDTH][TILE_WIDTH];
//
//    int bx = blockIdx.x;
//    int by = blockIdx.y;
//    int tx = threadIdx.x;
//    int ty = threadIdx.y;
//
//    int Row = (by * blockDim.y) + ty;
//    int Col = (bx * blockDim.x) + tx;
//
//    float Pvalue = 0;
//
//    for (int p = 0; p < (numAColumns - 1) / TILE_WIDTH + 1; p++) {
//        if (Row < numARows && p * TILE_WIDTH + tx < numAColumns) {
//            Ashared[ty][tx] = A[Row * numAColumns + (p * TILE_WIDTH + tx)];
//        }
//        else {
//            Ashared[ty][tx] = 0.0;
//        }
//        if (p * TILE_WIDTH + ty < numAColumns && Col < numBColumns) {
//            Bshared[ty][tx] = B[(p * TILE_WIDTH + ty) * numBColumns + Col];
//        }
//        else {
//            Bshared[ty][tx] = 0.0;
//        }
//        __syncthreads();
//
//        for (int i = 0; i < TILE_WIDTH; i++) {
//            Pvalue += Ashared[ty][i] * Bshared[i][tx];
//        }
//        __syncthreads();
//
//
//    }

//if (Row < numARows && Col < numBColumns) {
//    C[Row * numBColumns + Col] = Pvalue;
//}
//
//}

int main(int argc, char** argv) {
    wbArg_t args;
    float* hostA; // The A matrix
    float* hostB; // The B matrix
    float* hostC; // The output C matrix
    float* deviceA;
    float* deviceB;
    float* deviceC;
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows;    // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set
                     // this)

    hostC = NULL;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float*)wbImport(wbArg_getInputFile(args, 0), &numARows,
        &numAColumns);
    hostB = (float*)wbImport(wbArg_getInputFile(args, 1), &numBRows,
        &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;      // Rows of C = Rows of A
    numCColumns = numBColumns; // Columns of C = Columns of B

    //@@ Allocate the hostC matrix
    hostC = (float*)malloc(numCRows * numCColumns * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc((void**)&deviceA, numARows * numAColumns * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceB, numBRows * numBColumns * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceC, numCRows * numCColumns * sizeof(float)));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    //dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    //dim3 dimGrid((numCColumns + TILE_WIDTH - 1) / TILE_WIDTH, (numCRows + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 dimGrid((numCColumns - 1) / TILE_WIDTH + 1, (numCRows - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiplyShared << <dimGrid, dimBlock >> > (deviceA, deviceB, deviceC, numARows, numAColumns, numBColumns);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree(deviceA));
    wbCheck(hipFree(deviceB));
    wbCheck(hipFree(deviceC));

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
